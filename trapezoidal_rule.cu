/*
    By: Carrick McClain
    Sources:
        http://csweb.cs.wfu.edu
        https://stackoverflow.com
        http://www.cplusplus.com
        https://devtalk.nvidia.com
        https://docs.nvidia.com/cuda/cuda-c-programming-guide
*/

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>


using namespace std;

inline void gpu_handle_error( hipError_t err, const char* file, int line, int abort = 1 )
{
	if (err != hipSuccess)
	{
		fprintf (stderr, "gpu error: %s, %s, %d\n", hipGetErrorString (err), file, line);
		if (abort)
			exit (EXIT_FAILURE);
	}
}
#define gpu_err_chk(e) {gpu_handle_error( e, __FILE__, __LINE__ );}

/*
Integral Functions
You can replace any invoked math function with another.
    To test this, you can replace the function calls in the
    trapezoidal functions (host & device) with any of the others below.
I tried to implement these functions with functors, but they didn't work
as expected with device code.    */
float func_1a( float input )
{
    return 1/(1+input*input);
}
__device__ float func_1b( float input )
{
    return 1/(1+input*input);
}

// function 2 (host & gpu versions)
float func_2a( float input )
{
    return ((1.0*input*input) + (3.0*input*input) + 5.0);
}
__device__ float func_2b( float input )
{
    return ((1.0*input*input) + (3.0*input*input) + 5.0);
}

//function 3 (host & gpu versions)
float func_3a( float input )
{
    return ((2.0*input*input*input) / (5.0*input*input));
}
__device__ float func_3b( float input )
{
    return ((2.0*input*input*input) / (5.0*input*input));
}



// Serial trapezoidal rule function.
// Change around the commented lines to run it with other math functions.
float trapezoidal( float a, float b, float n )
{
    float delta = (b-a)/n;
    float s = func_1a(a) + func_1a(b);
    // float s = func_2a(a) + func_2a(b);
    // float s = func_3a(a) + func_3a(b);

    for( int i = 1; i < n; i++ )
    {
        s += 2.0*func_1a(a+i*delta);
        // s += 2.0*func_2a(a+i*delta);
        // s += 2.0*func_3a(a+i*delta);
    }
    return (delta/2)*s;
}

// Parallelized trapezoidal rule function.
// Change around the commented lines to run it with other math functions.
__global__ void trapezoidal_kernel( float a, float b, float n, float* d_output )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float delta = (b-a)/n;
    float s = a + (float)tid * delta;

    if( tid < n )
    {
        d_output[tid] = func_1b(s) + func_1b(s + delta);
        // d_output[tid] = func_2b(s) + func_2b(s + delta);
        // d_output[tid] = func_3b(s) + func_3b(s + delta);
    }
}

int main()
{
    // starts CUDA context, absorbs cost of startup
    // while starting, the program may seem to hang for a few seconds!
    // don't worry, it will work eventually.
    hipFree(0);    


    // initializations
    hipError_t err;
    float a = 0.0f;     // interval start
    float b = 1.0f;     // interval end
    int n = 10000;      // number of trapezoids
    float delta = (b-a)/n;
    float parallel_result = 0.0f;
    float* h_kernel_output = (float*)malloc(n * sizeof(float));
    float* d_kernel_output;
    cout.precision(5);

    // print out host function result
    cout << "Function 1: " << endl;
    cout << "Serial: Value of integral is " << trapezoidal(a, b, n) << endl;
    
    /* 
    Now the parallel part.
    The cudaMalloc was taking tons of time when I tested, not sure why.
    That's why I made the cudaFree(0) at the beginning.
    It absorbs the time cost of setting up the CUDA context,
        so the cudaMalloc() then takes much less time to execute.  */
    err = hipMalloc( (void**) &d_kernel_output, n * sizeof(float) );
    gpu_err_chk(err);
    err = hipMemcpy( d_kernel_output, h_kernel_output, n * sizeof(float), hipMemcpyHostToDevice );
    gpu_err_chk(err);
    
    
    // call kernel function
    dim3 dimGrid (40);      // threads/n -> 256 threads/block -> 40 blocks needed
    dim3 dimBlock (256); // 256
    trapezoidal_kernel<<<dimGrid, dimBlock>>>( a, b, n, d_kernel_output);
    err = hipGetLastError();
    gpu_err_chk(err);
    
    
    // copy data back from device
    err = hipMemcpy( h_kernel_output, d_kernel_output, n * sizeof(float), hipMemcpyDeviceToHost );
    gpu_err_chk(err);

    
    // get correct sum of trapezoid array
    for( int i=0; i<n; i++ )
    {
        parallel_result += h_kernel_output[i];
    }
    parallel_result *= delta/2.0;

    
    // print out device function result
    printf("Parallel: Value of integral is %6.4f\n", parallel_result);


    // free up memory
    free(h_kernel_output);
    hipFree(d_kernel_output);
    
    return 0;
}